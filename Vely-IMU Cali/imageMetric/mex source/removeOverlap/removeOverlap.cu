#include "hip/hip_runtime.h"
/* function for projecting lidar points
 *
 */

#include "../common.h"

__global__ void buildIndexKernel(const size_t imWidth,
									  const size_t imHeight,
									  const float* const xIn,
									  const float* const yIn,
									  const float* const d,
									  const size_t numPoints,
									  int* const idx){

	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;

	if(i >= numPoints){
		return;
	}

	int x = round(xIn[i]);
	int y = round(yIn[i]);

	if((x < 0) || (y < 0) || (x >= imWidth) || (y >= imHeight)){
		return;
	}
	else{
		int cI, check = 0;
		do {
			cI = idx[y + x*imHeight];
			if((cI < 0) || (d[cI] > d[i])){
				int check = atomicCAS(&idx[y + x*imHeight],cI,i);
			}
			else
			{
				break;
			}
		} while(cI != check);	
	}
}

__global__ void setValidKernel(bool* const valid,
									  const size_t idxSize,
									  const int* const idx){

	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;

	if(i >= idxSize){
		return;
	}
	if(idx[i] < 0){
		return;
	}
	valid[idx[i]] = true;
}

__global__ void setupIdxKernel(const size_t idxSize,
									  int* const idx){

	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;

	if(i >= idxSize){
		return;
	}
	idx[i] = -1;
}

void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
    //initialize the MathWorks GPU API.
    mxInitGPU();

    //read data
    mxGPUArray const * pointsMat = mxGPUCreateFromMxArray(prhs[0]);
    mxGPUArray const * distMat = mxGPUCreateFromMxArray(prhs[1]);
    size_t imWidth = ((uint32_T *) mxGetData(prhs[2]))[1];
    size_t imHeight = ((uint32_T *) mxGetData(prhs[2]))[0];
    size_t numPoints = mxGPUGetDimensions(pointsMat)[0];

	
    //get input pointers
    float* distPtr = (float*)(mxGPUGetDataReadOnly(distMat));

    float* xInPtr = (float*)(mxGPUGetDataReadOnly(pointsMat));
	float* yInPtr = &(xInPtr[numPoints]);
	
    //create output
	mwSize outSize[] = {numPoints,1};
    mxGPUArray* validMat = mxGPUCreateGPUArray(2, outSize, mxLOGICAL_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
	plhs[0] = mxGPUCreateMxArrayOnGPU(validMat);

	bool* validPtr = (bool*)(mxGPUGetData(validMat));

	//create idx matrix
	int *idxStore;
	CudaSafeCall(hipMalloc((void **)&idxStore, imWidth*imHeight*sizeof(int)));
	setupIdxKernel<<<gridSize(imWidth*imHeight), BLOCK_SIZE>>>(imWidth*imHeight, idxStore);
	CudaCheckError();
	
    //run and get ouputs
	buildIndexKernel<<<gridSize(numPoints), BLOCK_SIZE>>>(imWidth, imHeight, xInPtr, yInPtr, distPtr, numPoints, idxStore);
	CudaCheckError();
	setValidKernel<<<gridSize(imWidth*imHeight), BLOCK_SIZE>>>(validPtr, imWidth*imHeight, idxStore);
	CudaCheckError();
	
	//free idx matrix
	CudaSafeCall(hipFree(idxStore));
	
    //destroy reference structures
    mxGPUDestroyGPUArray(pointsMat);
    mxGPUDestroyGPUArray(distMat);
	mxGPUDestroyGPUArray(validMat);
}
